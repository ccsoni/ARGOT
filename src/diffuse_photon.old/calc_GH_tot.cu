#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "diffuse_photon.h"
#include "cuda_mem_space.h"

#ifndef TINY
#define TINY (1.0e-31)
#endif

__global__ void calc_GH_tot_kernel(struct radiation_mesh*, const struct step_func_factor* __restrict__);
__global__ void cuda_sum_GH(struct radiation_mesh*, const struct radiation_mesh* __restrict__);
__global__ void cuda_calc_GH(const struct radiation_mesh* __restrict__, struct fluid_mesh*);

extern "C" void calc_GH(struct cuda_mem_space* , struct cuda_diffuse_param*, struct cuda_param*, hipStream_t*);

extern "C"
void calc_GH_tot(struct cuda_mem_space *cuda_mem,
		 struct cuda_diffuse_param *cd_param,
		 hipStream_t strm, int device_id)
{
  dim3 block(NMESH_LOCAL/NMESH_PER_BLOCK_DMESH, 1, 1);
  dim3 thread(NMESH_PER_BLOCK_DMESH, 1, 1);
  
  CUDA_SAFE( hipSetDevice(device_id) );
  calc_GH_tot_kernel<<<block, thread, 0, strm>>>
    ( cd_param->rmesh, cd_param->step_fact );
}


__global__ void calc_GH_tot_kernel(struct radiation_mesh *rmesh, const struct step_func_factor* __restrict__ step_fact)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  
  float length_tot,I_tot;
  float I_seg_in_bar;

  length_tot  = rmesh[ix].length;

  I_tot    = rmesh[ix].IHI;
  I_seg_in_bar = I_tot/(length_tot*rmesh[ix].absorptionHI+TINY); 
  rmesh[ix].GHI_tot += I_seg_in_bar*step_fact->HI[0][0] + rmesh[ix].source_funcHI*step_fact->HI[0][1];
  rmesh[ix].HHI_tot += I_seg_in_bar*step_fact->HI[1][0] + rmesh[ix].source_funcHI*step_fact->HI[1][1];
  
#ifdef __HELIUM__
  I_tot    = rmesh[ix].IHeI;
  I_seg_in_bar = I_tot/(length_tot*rmesh[ix].absorptionHeI+TINY); 
  rmesh[ix].GHeI_tot += I_seg_in_bar*step_fact->HeI[0][0] + rmesh[ix].source_funcHeI*step_fact->HeI[0][1];
  rmesh[ix].HHeI_tot += I_seg_in_bar*step_fact->HeI[1][0] + rmesh[ix].source_funcHeI*step_fact->HeI[1][1];

  I_tot    = rmesh[ix].IHeII;
  I_seg_in_bar = I_tot/(length_tot*rmesh[ix].absorptionHeII+TINY); 
  rmesh[ix].GHeII_tot += I_seg_in_bar*step_fact->HeII[0][0] + rmesh[ix].source_funcHeII*step_fact->HeII[0][1];
  rmesh[ix].HHeII_tot += I_seg_in_bar*step_fact->HeII[1][0] + rmesh[ix].source_funcHeII*step_fact->HeII[1][1];
#endif //__HELIUM__ 
}


extern "C"
void calc_GH_sum(struct cuda_mem_space *cuda_mem,
		 struct cuda_diffuse_param *cd_param,
		 struct cuda_param *this_cuda)
{
  int idev;
  dim3 block, thread;
  int this_func_per_block = 1024;

  if(this_cuda->num_cuda_dev > 1) {
    block   = dim3(NMESH_LOCAL/this_func_per_block, 1, 1);
    thread  = dim3(this_func_per_block,1,1);

    for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
      CUDA_SAFE( hipSetDevice(idev) );
      CUDA_SAFE( hipDeviceSynchronize() );
    }
      
    if(this_cuda->num_cuda_dev == 2) {

      CUDA_SAFE( hipSetDevice(0) );
      cuda_sum_GH<<< block, thread, 0, this_cuda->strm[0] >>>
	(cd_param[0].rmesh, cd_param[1].rmesh);
      
      for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
	CUDA_SAFE( hipSetDevice(idev) );
	CUDA_SAFE( hipDeviceSynchronize() );
      }
      
    }else if(this_cuda->num_cuda_dev == 4) {
      
      for(int cloop=0; cloop<2; cloop++) {
	
#pragma omp parallel sections num_threads(2)
	{
#pragma omp section
	  {
	    CUDA_SAFE( hipSetDevice(0) );
	    cuda_sum_GH<<< block, thread, 0, this_cuda->strm[0] >>>
	      (cd_param[0].rmesh, cd_param[1].rmesh);
	  }      
#pragma omp section
	  {
	    CUDA_SAFE( hipSetDevice(2) );
	    cuda_sum_GH<<< block, thread, 0, this_cuda->strm[2] >>>
	      (cd_param[2].rmesh, cd_param[3].rmesh);
	  }  
	} //omp sections
	
	for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
	  CUDA_SAFE( hipSetDevice(idev) );
	  CUDA_SAFE( hipDeviceSynchronize() );
	}
	
	if(cloop==0) {

#pragma omp parallel sections num_threads(2)
	  {
#pragma omp section
	    {
	      CUDA_SAFE( hipSetDevice(0) );
	      CUDA_SAFE( hipMemcpyAsync( cd_param[3].rmesh, cd_param[0].rmesh,
					  sizeof(struct radiation_mesh)*NMESH_LOCAL,
					  hipMemcpyDefault, this_cuda->strm[0]) );
	    }
#pragma omp section
	    {
	      CUDA_SAFE( hipSetDevice(2) );
	      CUDA_SAFE( hipMemcpyAsync( cd_param[1].rmesh, cd_param[2].rmesh,
					  sizeof(struct radiation_mesh)*NMESH_LOCAL,
					  hipMemcpyDefault, this_cuda->strm[2]) );
	    }
	  } //omp sections
	  
	} else if(cloop==1) {

#pragma omp parallel sections num_threads(2)
	  {
#pragma omp section
	    {
	      CUDA_SAFE( hipSetDevice(0) );
	      CUDA_SAFE( hipMemcpyAsync( cd_param[1].rmesh, cd_param[0].rmesh,
					  sizeof(struct radiation_mesh)*NMESH_LOCAL,
					  hipMemcpyDefault, this_cuda->strm[0]) );
	    }
#pragma omp section
	    {
	      CUDA_SAFE( hipSetDevice(2) );
	      CUDA_SAFE( hipMemcpyAsync( cd_param[3].rmesh, cd_param[2].rmesh,
					  sizeof(struct radiation_mesh)*NMESH_LOCAL,
					  hipMemcpyDefault, this_cuda->strm[2]) );
	    }
	  } //omp sections
	} 
	

	for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
	  CUDA_SAFE( hipSetDevice(idev) );
	  CUDA_SAFE( hipDeviceSynchronize() );
	}
	
      } //cloop end
    } // if(cuda dev == 4)
  }  // if(cuda dev > 1)
  
  
  calc_GH(cuda_mem, cd_param, this_cuda, this_cuda->strm);
}


__global__ void cuda_sum_GH(struct radiation_mesh *rmesh1,
			    const struct radiation_mesh* __restrict__ rmesh2)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;

  rmesh1[ix].GHI_tot += rmesh2[ix].GHI_tot;
  rmesh1[ix].HHI_tot += rmesh2[ix].HHI_tot;
#ifdef __HELIUM__
  rmesh1[ix].GHeI_tot  += rmesh2[ix].GHeI_tot;
  rmesh1[ix].GHeII_tot += rmesh2[ix].GHeII_tot;
  rmesh1[ix].HHeI_tot  += rmesh2[ix].HHeI_tot;
  rmesh1[ix].HHeII_tot += rmesh2[ix].HHeII_tot;
#endif //__HELIUM__

}

extern "C"
void calc_GH(struct cuda_mem_space *cuda_mem,
	     struct cuda_diffuse_param *cd_param,
	     struct cuda_param *this_cuda,
	     hipStream_t *strm)
{
  dim3 block(NMESH_LOCAL/NMESH_PER_BLOCK_DMESH, 1, 1);
  dim3 thread(NMESH_PER_BLOCK_DMESH, 1, 1);  
  int idev;
 
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
    CUDA_SAFE( hipSetDevice(idev) );
    CUDA_SAFE( hipDeviceSynchronize() );
  }
  

#pragma omp parallel for num_threads(this_cuda->num_cuda_dev)
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
    CUDA_SAFE( hipSetDevice(idev) );
    cuda_calc_GH <<< block, thread, 0, strm[idev] >>>
      (cd_param[idev].rmesh, cuda_mem[idev].mesh_dev);
  }
  
  
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
    CUDA_SAFE( hipSetDevice(idev) );
    CUDA_SAFE( hipDeviceSynchronize() );
  }
  
}
  
__global__ void cuda_calc_GH(const struct radiation_mesh* __restrict__ rmesh, struct fluid_mesh *mesh)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  float r_n_ang = 1.0e0/N_ANG;
 
  ////Gamma_p += Gamma_d ,Heat_p += Heat_d 
  mesh[ix].prev_chem.GammaHI   += rmesh[ix].GHI_tot * r_n_ang;
  mesh[ix].prev_chem.HeatHI    += rmesh[ix].HHI_tot * r_n_ang; 
#ifdef __HELIUM__
  mesh[ix].prev_chem.GammaHeI  += rmesh[ix].GHeI_tot  * r_n_ang;
  mesh[ix].prev_chem.HeatHeI   += rmesh[ix].HHeI_tot  * r_n_ang;
  mesh[ix].prev_chem.GammaHeII += rmesh[ix].GHeII_tot * r_n_ang;
  mesh[ix].prev_chem.HeatHeII  += rmesh[ix].HHeII_tot * r_n_ang;
#endif //__HELIUM__
 }

