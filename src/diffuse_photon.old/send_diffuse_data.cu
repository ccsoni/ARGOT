#include <stdio.h>
#include <stdlib.h>

#include "diffuse_photon.h"
#include "cuda_mem_space.h"

extern "C" 
void send_diffuse_data(struct host_diffuse_param *hd_param,
		       struct cuda_diffuse_param *cd_param, struct cuda_param *this_cuda)
{
  static hipStream_t strm[NMAX_CUDA_DEV];
  int idev;

  /* Creating CUDA streams */
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++) {
    CUDA_SAFE( hipSetDevice(idev) );
    CUDA_SAFE( hipStreamCreate(&(strm[idev])) );
  }

  
  for(idev=0; idev < this_cuda->num_cuda_dev; idev++){
    CUDA_SAFE( hipSetDevice(idev) );
    
    CUDA_SAFE( hipMemcpyAsync( cd_param[idev].step_fact, hd_param->step_fact, 
				sizeof(struct step_func_factor), 
				hipMemcpyDefault, strm[idev]) );

    CUDA_SAFE( hipMemcpyAsync( cd_param[idev].angle, hd_param->angle, 
				sizeof(struct angle_info)*N_ANG, 
				hipMemcpyDefault, strm[idev]) );
  }
  

  /* Destroy CUDA streams */
  for(idev=0;idev<this_cuda->num_cuda_dev;idev++) {
    CUDA_SAFE( hipSetDevice(idev) );
    CUDA_SAFE( hipStreamDestroy(strm[idev]) );
  }
  
 
}

