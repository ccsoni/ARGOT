#include <stdio.h>
#include <stdlib.h>

#include "diffuse_photon.h"
#include "cuda_mem_space.h"

extern "C" 
void setup_ray_face_dev(struct ray_face *start_ray, struct cuda_param *this_cuda)
{
  int idev;

  for(idev=0; idev<this_cuda->num_cuda_dev; idev++) {
   
    CUDA_SAFE( hipSetDevice(idev) );
    
    CUDA_SAFE( hipMalloc((void **) &(start_ray[idev].xy),
			  sizeof(struct ray_info)*NMESH_MAX_FACE) );
    CUDA_SAFE( hipMalloc((void **) &(start_ray[idev].yz),
			  sizeof(struct ray_info)*NMESH_MAX_FACE) );
    CUDA_SAFE( hipMalloc((void **) &(start_ray[idev].zx),
			  sizeof(struct ray_info)*NMESH_MAX_FACE) );
  }
  
}

extern "C" 
void finalize_ray_face_dev(struct ray_face *start_ray, struct cuda_param *this_cuda)
{
  int idev;

  for(idev=0; idev<this_cuda->num_cuda_dev; idev++) {
   
    CUDA_SAFE( hipSetDevice(idev) );
    
    CUDA_SAFE( hipFree(start_ray[idev].xy) );
    CUDA_SAFE( hipFree(start_ray[idev].yz) );
    CUDA_SAFE( hipFree(start_ray[idev].zx) );
  }

}

extern "C" 
void send_ray_face(struct ray_face *ray1, struct ray_face *ray2,
		   hipStream_t cp_strm, int device_id)
{
  CUDA_SAFE( hipSetDevice(device_id) );

  CUDA_SAFE( hipMemcpyAsync(ray2->xy, ray1->xy, 
			     sizeof(struct ray_info)*NMESH_MAX_FACE,
			     hipMemcpyDefault, cp_strm) );
  
  CUDA_SAFE( hipMemcpyAsync(ray2->yz, ray1->yz, 
			     sizeof(struct ray_info)*NMESH_MAX_FACE,
			     hipMemcpyDefault, cp_strm) );
  
  CUDA_SAFE( hipMemcpyAsync(ray2->zx, ray1->zx, 
			     sizeof(struct ray_info)*NMESH_MAX_FACE,
			     hipMemcpyDefault, cp_strm) );

  CUDA_SAFE( hipStreamSynchronize(cp_strm) );
}

extern "C"
void cuda_set_device(int device_id)
{
  CUDA_SAFE( hipSetDevice(device_id) );
}
