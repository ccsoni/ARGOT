#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "diffuse_photon.h"
#include "cuda_mem_space.h"


#include "diffuse_chemistry.cu"

#ifndef TINY
#define TINY (1.0e-31)
#endif

extern "C" void merge_cuda_mem(struct cuda_mem_space*, struct cuda_param*, hipStream_t*);

__global__ void zero_set_rmesh_kernel(struct radiation_mesh*);
__global__ void calc_rmesh_data_kernel(const struct fluid_mesh* __restrict__, struct radiation_mesh*, 
				       const struct run_param* __restrict__);


extern "C" 
void zero_set_rmesh(struct radiation_mesh *rmesh,
		    hipStream_t strm, int device_id,struct ray_face *ray)
{
  int this_func_per_block = NMESH_PER_BLOCK_DMESH;

  dim3 block(NMESH_LOCAL/this_func_per_block,1,1);
  dim3 thread(this_func_per_block,1,1);
  
  CUDA_SAFE( hipSetDevice(device_id) );
  zero_set_rmesh_kernel<<<block, thread, 0, strm>>>
    ( rmesh );

}


__global__ void zero_set_rmesh_kernel(struct radiation_mesh *rmesh)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;

  rmesh[ix].length = 0.0;

  rmesh[ix].I_nu1 = 0.0;
#ifdef __HELIUM__
  rmesh[ix].I_nu2 = 0.0;
  rmesh[ix].I_nu3 = 0.0;
#ifdef __HELIUM_BB__
  rmesh[ix].I_nu4 = 0.0;
  rmesh[ix].I_nu5 = 0.0;
  rmesh[ix].I_nu6 = 0.0;
#endif
#endif //__HELIUM__
}


extern "C" 
void calc_rmesh_data(struct cuda_mem_space *cuda_mem,
		     struct cuda_diffuse_param *cd_param,
		     struct cuda_param *this_cuda)
{
  dim3 block(NMESH_LOCAL/NMESH_PER_BLOCK_DMESH,1,1);
  dim3 thread(NMESH_PER_BLOCK_DMESH,1,1);
  int idev;
  
  if(this_cuda->num_cuda_dev > 1) {
    merge_cuda_mem(cuda_mem, this_cuda, this_cuda->strm);
  }
  
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++) {
    
    CUDA_SAFE( hipSetDevice(idev) );
    calc_rmesh_data_kernel<<<block, thread, 0, this_cuda->strm[idev]>>>
      ( cuda_mem[idev].mesh_dev, cd_param[idev].rmesh,
	cuda_mem[idev].this_run_dev );
    
    CUDA_SAFE( hipStreamSynchronize(this_cuda->strm[idev]) );
  }
  
}


__global__ void calc_rmesh_data_kernel(const struct fluid_mesh* __restrict__ mesh,
				       struct radiation_mesh *rmesh,
				       const struct run_param* __restrict__ this_run)
{
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;

  float csecHI   = csectHI_dev(HI_LYMAN_LIMIT+1.0e-7);      //13.6eV
#ifdef __HELIUM__
  float csecHeI  = csectHeI_dev(HeI_LYMAN_LIMIT+1.0e-7);    //24.6eV
  float csecHeII = csectHeII_dev(HeII_LYMAN_LIMIT+1.0e-7);  //54.4eV
#endif //__HELIUM__

  float wmol, temper;
  double emission; 
  double nH,nHI,nHII,ne;
    
  wmol   = WMOL(mesh[ix].prev_chem);
  temper = mesh[ix].prev_uene * this_run->uenetok * wmol;
  if(temper < 1.0) temper = 1.0;
  
  nH   = mesh[ix].dens * this_run->denstonh;
#ifdef __COSMOLOGICAL__
  nH  /= CUBE(this_run->anow); 
#endif

  nHI  = mesh[ix].prev_chem.fHI  * nH;
  nHII = mesh[ix].prev_chem.fHII * nH;
  ne   = nHII;
#ifdef __HELIUM__
  double nHeI,nHeII,nHeIII;
  nHeI  = mesh[ix].prev_chem.fHeI  * HELIUM_FACT*nH;
  nHeII = mesh[ix].prev_chem.fHeII * HELIUM_FACT*nH;
  nHeIII= mesh[ix].prev_chem.fHeIII* HELIUM_FACT*nH;
  
  ne += nHeII + 2.0e0*nHeIII;
#endif  

  /* absorption */
  double absorptionHI_nu1 = csecHI*nHI;
  rmesh[ix].absorption_nu1 = absorptionHI_nu1 + TINY;   
  
#ifdef __HELIUM__
  double absorptionHeI_nu2, absorptionHeII_nu3;
  absorptionHeI_nu2  = csecHeI*nHeI;
  absorptionHeII_nu3 = csecHeII*nHeII;
  
  rmesh[ix].absorption_nu2 = absorptionHI_nu1*RCROSS_HI_nu2 + absorptionHeI_nu2 + TINY;
  rmesh[ix].absorption_nu3 = absorptionHI_nu1*RCROSS_HI_nu3 + absorptionHeI_nu2*RCROSS_HeI_nu3 + absorptionHeII_nu3 + TINY;
  
#ifdef __HELIUM_BB__
  rmesh[ix].absorption_nu4 = absorptionHI_nu1*RCROSS_HI_nu4 + TINY;
  rmesh[ix].absorption_nu5 = absorptionHI_nu1*RCROSS_HI_nu5 + TINY;
  rmesh[ix].absorption_nu6 = absorptionHI_nu1*RCROSS_HI_nu6 + absorptionHeI_nu2*RCROSS_HeI_nu6 + TINY;
#endif //__HELIUM_BB__    
#endif //__HELIUM__
    
  /* emissivity and source function */
  emission = (k02_A(temper)-k02_B(temper))*ne*nHII * nuLeV*HI_LYMAN_LIMIT*eV_to_erg/(4.0*PI);
  rmesh[ix].source_func_nu1 = emission/rmesh[ix].absorption_nu1;
  
#ifdef __HELIUM__
  emission = (k04_A(temper)-k04_B(temper))*ne*nHeII * nuLeV*HeI_LYMAN_LIMIT*eV_to_erg/(4.0*PI);
  rmesh[ix].source_func_nu2 = emission/rmesh[ix].absorption_nu2;
  
  emission = (k06_A(temper)-k06_B(temper))*ne*nHeIII * nuLeV*HeII_LYMAN_LIMIT*eV_to_erg/(4.0*PI);
  rmesh[ix].source_func_nu3 = emission/rmesh[ix].absorption_nu3;
  
#ifdef __HELIUM_BB__
  emission = 0.75*k04_B(temper)*ne*nHeII * nuLeV*HeI_BBT_ENG*eV_to_erg/(4.0*PI);
  rmesh[ix].source_func_nu4 = emission/rmesh[ix].absorption_nu4;
  
  emission = (1.0/6.0)*k04_B(temper)*ne*nHeII * nuLeV*HeI_BBS_ENG*eV_to_erg/(4.0*PI);
  rmesh[ix].source_func_nu5 = emission/rmesh[ix].absorption_nu5;

  emission = k06_B(temper)*ne*nHeIII * nuLeV*HeII_BB_ENG*eV_to_erg/(4.0*PI);
  rmesh[ix].source_func_nu6 = emission/rmesh[ix].absorption_nu6;
#endif //__HELIUM_BB__    
#endif //__HELIUM__

  ///zero_set
  rmesh[ix].GHI_tot   = 0.0e0;
  rmesh[ix].HHI_tot   = 0.0e0;
#ifdef __HELIUM__
  rmesh[ix].GHeI_tot  = 0.0e0;
  rmesh[ix].HHeI_tot  = 0.0e0;
  rmesh[ix].GHeII_tot = 0.0e0;
  rmesh[ix].HHeII_tot = 0.0e0;
#endif //__HELIUM__
}



extern "C"
void merge_cuda_mem(struct cuda_mem_space *cuda_mem, 
		struct cuda_param *this_cuda,
		hipStream_t *strm)
{
  size_t mesh_s = sizeof(struct fluid_mesh)*NMESH_LOCAL/this_cuda->num_cuda_dev;
  
  int idev;
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
    CUDA_SAFE( hipSetDevice(idev) );
    CUDA_SAFE( hipDeviceSynchronize() );
  }
  
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
    
    long begin_address;
    int tid;   
    
    CUDA_SAFE( hipSetDevice(idev) );
    
    begin_address = idev * (NMESH_LOCAL / this_cuda->num_cuda_dev);
    
    for(tid=0;tid<this_cuda->num_cuda_dev;tid++){
      if(idev==tid) continue;
      
      CUDA_SAFE( hipMemcpyAsync(&cuda_mem[tid].mesh_dev[begin_address], &cuda_mem[idev].mesh_dev[begin_address],
				 mesh_s, hipMemcpyDefault, strm[idev]) );
      
    }
    
  }
  
  for(idev=0; idev<this_cuda->num_cuda_dev; idev++){
    CUDA_SAFE( hipSetDevice(idev) );
    CUDA_SAFE( hipDeviceSynchronize() );
  }
  
}

