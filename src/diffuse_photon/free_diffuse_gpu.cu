#include <stdio.h>
#include <stdlib.h>

#include "diffuse_photon.h"
#include "cuda_mem_space.h"

extern "C" 
void free_diffuse_gpu(struct cuda_diffuse_param *cd_param, struct cuda_param *this_cuda)
{
  int did;
  for(did=0; did < this_cuda->num_cuda_dev; did++){
    CUDA_SAFE( hipSetDevice(did) );
    
    CUDA_SAFE( hipFree(cd_param[did].step_fact) );
    CUDA_SAFE( hipFree(cd_param[did].angle) );
    CUDA_SAFE( hipFree(cd_param[did].rmesh) );
 
  }

#if 0
  if(this_cuda->num_cuda_dev==2) { 
    CUDA_SAFE( hipSetDevice(0) );
    CUDA_SAFE( hipDeviceDisablePeerAccess(1) );
    CUDA_SAFE( hipSetDevice(1) );
    CUDA_SAFE( hipDeviceDisablePeerAccess(0) );    
  }

  if(this_cuda->num_cuda_dev==4) { 
    CUDA_SAFE( hipSetDevice(0) );
    CUDA_SAFE( hipDeviceDisablePeerAccess(1) );     
    CUDA_SAFE( hipSetDevice(1) );
    CUDA_SAFE( hipDeviceDisablePeerAccess(0) );    

    CUDA_SAFE( hipSetDevice(2) );
    CUDA_SAFE( hipDeviceDisablePeerAccess(3) );     
    CUDA_SAFE( hipSetDevice(3) );
    CUDA_SAFE( hipDeviceDisablePeerAccess(2) ); 
  }
#endif
}

