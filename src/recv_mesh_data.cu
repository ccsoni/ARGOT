#include <stdio.h>
#include <assert.h>
#include <stdint.h>

#include "fluid.h"
#include "cuda_mem_space.h"
#include "run_param.h"

extern "C"
void recv_mesh_data(struct fluid_mesh *mesh, struct cuda_mem_space *cuda_mem, 
		    struct cuda_param *this_cuda, struct run_param *this_run)
{
  hipError_t err;

  int idev;

  for(idev=0;idev<this_cuda->num_cuda_dev;idev++) {
    hipSetDevice(idev);
    uint64_t offset;

    offset = idev*this_cuda->cuda_nblock*NMESH_PER_BLOCK;

    err = hipMemcpyAsync(mesh+offset, cuda_mem[idev].mesh_dev+offset,
			  sizeof(struct fluid_mesh)*this_cuda->nmesh_per_dev,
			  hipMemcpyDeviceToHost, this_cuda->strm[idev]);
    assert(err == hipSuccess);
  }

  for(idev=0;idev<this_cuda->num_cuda_dev;idev++) {
    hipSetDevice(idev);
    hipStreamSynchronize(this_cuda->strm[idev]);
  }

}
