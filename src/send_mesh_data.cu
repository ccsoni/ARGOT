#include <stdio.h>
#include <assert.h>

#include "fluid.h"
#include "cuda_mem_space.h"
#include "run_param.h"

extern "C"
void send_mesh_data(struct fluid_mesh *mesh, struct cuda_mem_space *cuda_mem, 
                    struct cuda_param *this_cuda, struct run_param *this_run)
{
  hipError_t err;

  int idev;

  for(idev=0;idev<this_cuda->num_cuda_dev;idev++) {
    hipSetDevice(idev);
    err = hipMemcpyAsync(cuda_mem[idev].mesh_dev, mesh, 
			  sizeof(struct fluid_mesh)*NMESH_LOCAL, 
			  hipMemcpyHostToDevice, this_cuda->strm[idev]);
    assert(err == hipSuccess);
  }

  for(idev=0;idev<this_cuda->num_cuda_dev;idev++) {
    hipSetDevice(idev);
    hipStreamSynchronize(this_cuda->strm[idev]);
  }

}
